#include "hip/hip_runtime.h"
//
// Created by Noah Kubli on 12.03.2024.
//
#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/partition.h>
#include <thrust/sequence.h>

#include "cstone/cuda/cuda_utils.cuh"
#include "cstone/findneighbors.hpp"
#include "cstone/traversal/find_neighbors.cuh"
#include "sph/util/device_math.cuh"

#include "cstone/sfc/box.hpp"
#include "cstone/tree/definitions.h"

#include "sph/particles_data.hpp"

#include "accretion_gpu.hpp"
#include "star_data.hpp"
#include "hip/hip_runtime.h"

static __device__ double   dev_accr_mass;
static __device__ double   dev_accr_mom_x;
static __device__ double   dev_accr_mom_y;
static __device__ double   dev_accr_mom_z;
static __device__ unsigned dev_n_removed;
static __device__ unsigned dev_n_accreted;

using cstone::TravConfig;

template<typename T1, typename Th, typename Tremove, typename T2, typename Tm, typename Tv>
__global__ void computeAccretionConditionKernel(size_t first, size_t last, const T1* x, const T1* y, const T1* z,
                                                const Th* h, Tremove* remove, const Tm* m, const Tv* vx, const Tv* vy,
                                                const Tv* vz, T2 star_x, T2 star_y, T2 star_z, T2 star_size2,
                                                T2 removal_limit_h)
{
    cstone::LocalIndex i = first + blockDim.x * blockIdx.x + threadIdx.x;
    double             accr_mass{};
    double             accr_mom_x{};
    double             accr_mom_y{};
    double             accr_mom_z{};
    unsigned           accreted{};
    unsigned           removed{};

    if (i >= last) {}
    else
    {
        const double dx    = x[i] - star_x;
        const double dy    = y[i] - star_y;
        const double dz    = z[i] - star_z;
        const double dist2 = dx * dx + dy * dy + dz * dz;

        if (dist2 < star_size2)
        {
            // Accrete on star
            remove[i]  = cstone::removeKey<Tremove>::value;
            accr_mass  = m[i];
            accr_mom_x = m[i] * vx[i];
            accr_mom_y = m[i] * vy[i];
            accr_mom_z = m[i] * vz[i];
            accreted   = 1;
        }
        else if (h[i] > removal_limit_h)
        {
            // Remove from system
            remove[i] = cstone::removeKey<Tremove>::value;
            removed   = 1;
        }
    }
    typedef hipcub::BlockReduce<double, TravConfig::numThreads> BlockReduceDouble;

    __shared__ typename BlockReduceDouble::TempStorage temp_accr_mass;

    double block_accr_mass = BlockReduceDouble(temp_accr_mass).Reduce(accr_mass, hipcub::Sum());

    __shared__ typename BlockReduceDouble::TempStorage temp_accr_mom_x, temp_accr_mom_y, temp_accr_mom_z;

    double block_accr_mom_x = BlockReduceDouble(temp_accr_mom_x).Reduce(accr_mom_x, hipcub::Sum());
    double block_accr_mom_y = BlockReduceDouble(temp_accr_mom_y).Reduce(accr_mom_y, hipcub::Sum());
    double block_accr_mom_z = BlockReduceDouble(temp_accr_mom_z).Reduce(accr_mom_z, hipcub::Sum());

    typedef hipcub::BlockReduce<unsigned, TravConfig::numThreads> BlockReduceUnsigned;

    __shared__ typename BlockReduceUnsigned::TempStorage temp_storage_n_rem, temp_storage_n_accr;

    unsigned block_n_removed  = BlockReduceUnsigned(temp_storage_n_rem).Reduce(removed, hipcub::Sum());
    unsigned block_n_accreted = BlockReduceUnsigned(temp_storage_n_accr).Reduce(accreted, hipcub::Sum());

    __syncthreads();

    if (threadIdx.x == 0)
    {
        atomicAdd(&dev_accr_mass, block_accr_mass);
        atomicAdd(&dev_accr_mom_x, block_accr_mom_x);
        atomicAdd(&dev_accr_mom_y, block_accr_mom_y);
        atomicAdd(&dev_accr_mom_z, block_accr_mom_z);
        atomicAdd(&dev_n_removed, block_n_removed);
        atomicAdd(&dev_n_accreted, block_n_accreted);
    }
}

template<typename Dataset, typename StarData>
void computeAccretionConditionGPU(size_t first, size_t last, Dataset& d, StarData& star)
{
    cstone::LocalIndex numParticles = last - first;
    unsigned           numThreads   = 256;
    unsigned           numBlocks    = (numParticles + numThreads - 1) / numThreads;

    double   zero   = 0.;
    unsigned zero_s = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(dev_accr_mass), &zero, sizeof(zero));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_accr_mom_x), &zero, sizeof(zero));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_accr_mom_y), &zero, sizeof(zero));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_accr_mom_z), &zero, sizeof(zero));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_n_removed), &zero_s, sizeof(zero_s));
    hipMemcpyToSymbol(HIP_SYMBOL(dev_n_accreted), &zero_s, sizeof(zero_s));

    computeAccretionConditionKernel<<<numBlocks, numThreads>>>(
        first, last, rawPtr(d.devData.x), rawPtr(d.devData.y), rawPtr(d.devData.z), rawPtr(d.devData.h),
        rawPtr(d.devData.keys), rawPtr(d.devData.m), rawPtr(d.devData.vx), rawPtr(d.devData.vy), rawPtr(d.devData.vz),
        star.position[0], star.position[1], star.position[2], star.inner_size * star.inner_size, star.removal_limit_h);
    checkGpuErrors(hipGetLastError());
    checkGpuErrors(hipDeviceSynchronize());

    double   m_accr_ret;
    double   px_accr_ret;
    double   py_accr_ret;
    double   pz_accr_ret;
    unsigned n_removed;
    unsigned n_accr;

    hipMemcpyFromSymbol(&m_accr_ret, HIP_SYMBOL(dev_accr_mass), sizeof(m_accr_ret));
    hipMemcpyFromSymbol(&px_accr_ret, HIP_SYMBOL(dev_accr_mom_x), sizeof(px_accr_ret));
    hipMemcpyFromSymbol(&py_accr_ret, HIP_SYMBOL(dev_accr_mom_y), sizeof(py_accr_ret));
    hipMemcpyFromSymbol(&pz_accr_ret, HIP_SYMBOL(dev_accr_mom_z), sizeof(pz_accr_ret));
    hipMemcpyFromSymbol(&n_removed, HIP_SYMBOL(dev_n_removed), sizeof(n_removed));
    hipMemcpyFromSymbol(&n_accr, HIP_SYMBOL(dev_n_accreted), sizeof(n_accr));

    star.m_accreted_local    = m_accr_ret;
    star.p_accreted_local[0] = px_accr_ret;
    star.p_accreted_local[1] = py_accr_ret;
    star.p_accreted_local[2] = pz_accr_ret;
    star.n_removed_local     = n_removed;
    star.n_accreted_local    = n_accr;
}

template void computeAccretionConditionGPU(size_t, size_t, sphexa::ParticlesData<cstone::GpuTag>&, StarData&);
